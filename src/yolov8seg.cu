#include "hip/hip_runtime.h"
#include "yolov8seg.h"
// #include "postprocess.cu"

#include <fstream>
#include <cmath>

#include "NvInferVersion.h"
#include "NvOnnxParser.h"
#include "hip/hip_runtime.h"

#include <opencv2/highgui.hpp>
#include <opencv2/dnn.hpp>
#include <opencv2/imgproc.hpp>

#define BLOCK_SIZE 32

__global__ void gpu_matrix_mult(float *a, float *b, float *c, int m, int n, int k, int iVal, int numClasses)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[iVal + (4 + numClasses + i) * 8400] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

float sigmoid(float num)
{
    return (float)((1 / (1 + exp(-num))) > 0.5);
}

Detection::Detection(int classId, float conf,
                     cv::Rect boundingBox, cv::Mat mask)
    : classId_(classId), conf_(conf), boundingBox_(boundingBox), mask_(mask) {}

int Detection::classId() { return classId_; }

float Detection::confidence() { return conf_; }

cv::Rect Detection::bbox() { return boundingBox_; }

cv::Mat Detection::mask() { return mask_; }

int YoloV8Detector::numClasses() { return numClasses_; }

YoloV8Detector::YoloV8Detector(std::string filepath,
                               float conf_threshold,
                               float iou_threshold)
    : confThreshold_(conf_threshold), iouThreshold_(iou_threshold)
{
    std::ifstream file(filepath, std::ios::binary | std::ios::ate);
    if (!file.good())
    {
        std::string errorMsg = "File " + filepath + " does not exist";

        throw std::runtime_error(errorMsg);
    }

    // printf()

    Logger logger;

    std::ifstream engineFile(
        filepath.substr(0, filepath.find_last_of('.')) + ".trt", std::ios::binary | std::ios::ate);

    if (engineFile.good())
    {
        printf("TRT engine file detected\n");
        // engineFile = std::move(file);
    }

    else if (filepath.substr(filepath.size() - 5, filepath.size()) == ".onnx")
    {
        // Convert

        printf("Engine file not detected: creating now...\n");
        nvinfer1::IBuilder *builder = nvinfer1::createInferBuilder(logger);

        auto explicitBatch =
            1U << static_cast<uint32_t>(
                nvinfer1::NetworkDefinitionCreationFlag::kEXPLICIT_BATCH);

        // auto network = builder->createNetworkV2(0);

        std::unique_ptr<nvinfer1::INetworkDefinition> network(
            builder->createNetworkV2(explicitBatch));

        if (!network)
        {
            throw std::runtime_error("Failed to build network");
        }

        std::streamsize size = file.tellg();
        file.seekg(0, std::ios::beg);
        std::vector<char> buffer(size);
        if (!file.read(buffer.data(), size))
        {
            throw std::runtime_error("Error, unable to read onnx file");
        }
        file.close();

        auto parser = std::unique_ptr<nvonnxparser::IParser>(
            nvonnxparser::createParser(*network, logger));
        if (!parser)
        {
            throw std::runtime_error("Failed to create parser");
        }

        auto parsed = parser->parseFromFile(filepath.c_str(), 0);
        if (!parsed)
        {
            throw std::runtime_error("Failed to parse onnx file");
        }

        // printf("Errors %d\n", parser->getNbErrors());

        std::unique_ptr<nvinfer1::IBuilderConfig> config(
            builder->createBuilderConfig());
        if (!config)
        {
            throw std::runtime_error("Failed to build config");
        }

        // printf("%d\n", network->getNbOutputs());

        // Has to be raw or else it won't compile
        nvinfer1::IOptimizationProfile *optProfile =
            builder->createOptimizationProfile();
        const auto input = network->getInput(0);
        const auto inputName = input->getName();
        const auto inputDims = input->getDimensions();
        int32_t inputC = inputDims.d[1];
        int32_t inputH = inputDims.d[2];
        int32_t inputW = inputDims.d[3];

        optProfile->setDimensions(inputName, nvinfer1::OptProfileSelector::kMIN,
                                  nvinfer1::Dims4(1, inputC, inputH, inputW));
        optProfile->setDimensions(inputName, nvinfer1::OptProfileSelector::kOPT,
                                  nvinfer1::Dims4(1, inputC, inputH, inputW));
        optProfile->setDimensions(inputName, nvinfer1::OptProfileSelector::kMAX,
                                  nvinfer1::Dims4(1, inputC, inputH, inputW));

        config->addOptimizationProfile(optProfile);

        // Do we want to add functionality to change the percision of the NN?
        // config.setFlag(nvinfer1::BuilderFlag::KFP16);

        // hipStream_t profileStream;
        // hipError_t ret = hipStreamCreate(&profileStream);
        // if (ret != hipSuccess) {
        //     throw std::runtime_error("Failed to create profile stream: " +
        //                              std::string(hipGetErrorName(ret)) +
        //                              '\n' +
        //                              std::string(hipGetErrorString(ret)));
        // }
        // config->setProfileStream(profileStream);
        std::unique_ptr<nvinfer1::IHostMemory> plan(
            builder->buildSerializedNetwork(*network, *config));
        if (!plan)
        {
            throw std::runtime_error("Failed to build engine");
        }
        std::ofstream outfile(filepath.substr(0, filepath.size() - 5) + ".trt",
                              std::ofstream::binary);
        outfile.write(reinterpret_cast<const char *>(plan->data()),
                      plan->size());
        outfile.close();

        printf("Successfully created engine file\n");

        // hipStreamDestroy(profileStream);

        engineFile =
            std::ifstream(filepath.substr(0, filepath.size() - 5) + ".trt",
                          std::ios::binary | std::ios::ate);
    }
    else
    {
        std::string errorMsg = "Invalid file type: Expects .onnx or .trt";

        throw std::runtime_error(errorMsg);
    }

    std::streamsize engineFileSize = engineFile.tellg();
    engineFile.seekg(0, std::ios::beg);
    std::vector<char> engineFileBuffer(engineFileSize);
    if (!engineFile.read(engineFileBuffer.data(), engineFileSize))
    {
        auto msg = "Error, unable to read engine file";
        throw std::runtime_error(msg);
    }
    // printf("Size: %d\n", engineFileBuffer.size());

    // printf("creating runtime\n");
    runtime = nvinfer1::createInferRuntime(logger);

    // printf("building runtime\n");
    nvinfer1::ICudaEngine *engine = (runtime->deserializeCudaEngine(
        engineFileBuffer.data(), engineFileBuffer.size()));
    // printf("built runtime\n");
    if (!engine)
    {
        throw std::runtime_error("Failed to create engine");
    }

    // printf("creating context\n");
    context = engine->createExecutionContext();
    // printf("created context\n");

    hipError_t ret = hipStreamCreate(&contextStream);

    if (ret != hipSuccess)
    {
        throw std::runtime_error("Failed to create context stream: " +
                                 std::string(hipGetErrorName(ret)) + '\n' +
                                 std::string(hipGetErrorString(ret)));
    }

    numClasses_ = engine->getTensorShape("output0").d[1] - 4 - 32;
    hipDeviceSynchronize();

    output0Copy = new float[8400 * (numClasses_ + 4 + 32)];
    output1Copy = new float[160 * 160 * 32];

    ret = hipMallocManaged((void **)(&input), 640 * 640 * 3 * sizeof(float));

    if (ret != hipSuccess)
    {
        throw std::runtime_error("Failed to malloc input: " +
                                 std::string(hipGetErrorName(ret)) + '\n' +
                                 std::string(hipGetErrorString(ret)));
    }
    ret = hipMallocManaged((void **)(&output0), 8400 * (numClasses_ + 4 + 32) * sizeof(float));

    // printf("Num classes: %d\n", numClasses_);

    if (ret != hipSuccess)
    {
        throw std::runtime_error("Failed to malloc output0: " +
                                 std::string(hipGetErrorName(ret)) + '\n' +
                                 std::string(hipGetErrorString(ret)));
    }
    ret = hipMallocManaged((void **)(&output1), 160 * 160 * 32 * sizeof(float));

    if (ret != hipSuccess)
    {
        throw std::runtime_error("Failed to malloc output1: " +
                                 std::string(hipGetErrorName(ret)) + '\n' +
                                 std::string(hipGetErrorString(ret)));
    }
    ret = hipMallocManaged((void **)(&maskWeights), 32 * sizeof(float));

    if (ret != hipSuccess)
    {
        throw std::runtime_error("Failed to malloc maskWeights: " +
                                 std::string(hipGetErrorName(ret)) + '\n' +
                                 std::string(hipGetErrorString(ret)));
    }
    ret = hipMallocManaged((void **)(&mask), 160 * 160 * sizeof(float));

    if (ret != hipSuccess)
    {
        throw std::runtime_error("Failed to malloc mask: " +
                                 std::string(hipGetErrorName(ret)) + '\n' +
                                 std::string(hipGetErrorString(ret)));
    }

    bool worked = context->setInputTensorAddress("images", input);
    if (!worked)
    {
        printf("Failed to set images\n");
    }
    worked = context->setTensorAddress("output0", output0);
    if (!worked)
    {
        printf("Failed to set output0\n");
    }
    worked = context->setTensorAddress("output1", output1);
    if (!worked)
    {
        printf("Failed to set output1\n");
    }
}

YoloV8Detector::~YoloV8Detector()
{
    // context->destroy();
    hipStreamDestroy(contextStream);
    hipFree(input);
    hipFree(output0);
    hipFree(output1);
    if (runtime != nullptr)
    {
        runtime = nullptr;
    }

    delete output0Copy;
    delete output1Copy;
}

std::vector<Detection> YoloV8Detector::runDetection(cv::Mat &image)
{
    using namespace std::chrono;

    cv::Mat resized, preprocessed;

    float aspectRatio, colRatio, rowRatio;
    int topBorder, sideBorder;
    bool widthLarger = false;

    if (image.cols > image.rows)
    {
        aspectRatio = (float)image.rows / image.cols;
        widthLarger = true;
    }
    else
    {
        aspectRatio = (float)image.cols / image.rows;
    }

    cv::Size newSize;
    if (widthLarger)
    {
        newSize = cv::Size(640, 640 * aspectRatio);
        topBorder = (640. - newSize.height) / 2;
        sideBorder = 0;
        colRatio = (float)image.cols / 640.;
        rowRatio = (float)image.rows / 640. / aspectRatio;
    }
    else
    {
        newSize = cv::Size(640 * aspectRatio, 640);
        topBorder = 0;
        sideBorder = (640. - newSize.width) / 2;
        colRatio = (float)image.cols / 640. / aspectRatio;
        rowRatio = (float)image.rows / 640.;
    }

    cv::resize(image, resized, newSize, 0, 0, cv::INTER_LINEAR);
    cv::copyMakeBorder(resized, preprocessed, topBorder, topBorder, sideBorder, sideBorder, cv::BORDER_CONSTANT, cv::Scalar(0, 0, 0));

    preprocessed.convertTo(preprocessed, CV_32FC3);
    preprocessed /= 255.;
    // cv::imshow("preprocessed", preprocessed);
    // cv::waitKey(0);
    printf("preprocessed size: %d %d\n", preprocessed.rows, preprocessed.cols);

    cv::Mat blob = cv::dnn::blobFromImage(preprocessed);
    auto start = high_resolution_clock::now();
    hipError_t ret = hipMemcpy(input, blob.data, 640 * 640 * 3 * sizeof(float), hipMemcpyHostToDevice);
    auto firstMemcpy = high_resolution_clock::now();

    if (ret != hipSuccess)
    {
        throw std::runtime_error("Failed to memcpy to input: " +
                                 std::string(hipGetErrorName(ret)) + '\n' +
                                 std::string(hipGetErrorString(ret)));
    }

    bool success = context->enqueueV3(contextStream);

    hipStreamSynchronize(contextStream);
    auto enqueued = high_resolution_clock::now();

    if (!success)
    {
        throw std::runtime_error("Failed to run inference\n");
    }

    ret = hipMemcpy(output0Copy, output0, 8400 * (numClasses_ + 4 + 32) * 4,
                     hipMemcpyDeviceToHost);

    auto output0Memcpy = high_resolution_clock::now();

    if (ret != hipSuccess)
    {
        throw std::runtime_error("Failed to memcpy to output0Copy: " +
                                 std::string(hipGetErrorName(ret)) + '\n' +
                                 std::string(hipGetErrorString(ret)));
    }

    std::vector<std::vector<cv::Rect>> classDetections(numClasses_);
    std::vector<std::vector<float>> classScores(numClasses_);
    std::vector<std::vector<int>> iValues(numClasses_);
    std::vector<Detection> detections;
    for (int i = 0; i < 8400; i++)
    {
        // Detection detection;
        float maxConf = -1;
        int classId = 0;
        for (int j = 0; j < numClasses_; j++)
        {
            float conf = output0Copy[i + (j + 4) * 8400];
            if (conf > maxConf)
            {
                classId = j;
                maxConf = conf;
            }
        }
        if (maxConf > confThreshold_)
        {
            // printf("%f\n", )
            float normBbox[4];
            normBbox[0] = output0Copy[i] * colRatio - sideBorder * 2;
            normBbox[1] = output0Copy[i + 8400] * rowRatio - topBorder * 2;
            normBbox[2] = output0Copy[i + 2 * 8400] * colRatio;
            normBbox[3] = output0Copy[i + 3 * 8400] * rowRatio;
            cv::Rect bbox = cv::Rect(normBbox[0] - normBbox[2] / 2,
                                     normBbox[1] - normBbox[3] / 2, normBbox[2],
                                     normBbox[3]);
            classDetections[classId].push_back(bbox);
            classScores[classId].push_back(maxConf);
            iValues[classId].push_back(i);
        }
    }

    for (int i = 0; i < numClasses_; i++)
    {
        std::vector<int> indices;
        cv::dnn::NMSBoxes(classDetections[i], classScores[i], confThreshold_, iouThreshold_, indices);
        for (int index : indices)
        {
            cv::Mat croppedMask = cv::Mat::zeros(preprocessed.size(), CV_32FC1);
            cv::Mat mask = calculateMask(iValues[i][index]);
            if (widthLarger)
            {
                cv::resize(mask, mask, cv::Size(image.cols, image.cols));
            }
            else
            {
                cv::resize(mask, mask, cv::Size(image.rows, image.rows));
            }
            croppedMask(classDetections[i][index]) = mask(cv::Rect(classDetections[i][index].x - sideBorder, classDetections[i][index].y - topBorder, classDetections[i][index].width, classDetections[i][index].height));
            croppedMask.convertTo(croppedMask, CV_8UC1);
            cv::cvtColor(croppedMask, croppedMask, cv::COLOR_GRAY2BGR);
            detections.push_back(Detection(i, classScores[i][index], classDetections[i][index], croppedMask));
        }
    }

    auto postProcess = high_resolution_clock::now();

    // printf("Enqueue time: %f\n", (float)duration_cast<nanoseconds>(enqueued - start).count() / 1e6);
    // printf("output0 memcpy time: %f\n", (float)duration_cast<nanoseconds>(output0Memcpy - enqueued).count() / 1e6);
    // printf("postprocess time: %f\n", (float)duration_cast<nanoseconds>(postProcess - output0Memcpy).count() / 1e6);

    // for (auto classDets : classDetections)
    // {
    //     for (auto detection : classDets)
    //     {
    //         detections.push_back(detection);
    //     }
    // }

    return detections;
}

cv::Mat YoloV8Detector::calculateMask(int iVal)
{
    unsigned int gridRows = (1 + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int gridCols = (160 * 160 + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(gridCols, gridRows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(output0, output1, mask, 1, 32, 160 * 160, iVal, numClasses_);
    hipDeviceSynchronize();
    cv::Mat cpuMask(160, 160, CV_32FC1);

    hipMemcpy(cpuMask.data, mask, 160 * 160 * sizeof(float), hipMemcpyDeviceToHost);

    // double maxVal;
    // cv::minMaxLoc(cpuMask, nullptr, &maxVal);

    // printf("Max value: %f\n", maxVal);
    std::transform(cpuMask.begin<float>(), cpuMask.end<float>(), cpuMask.begin<float>(), sigmoid);

    return cpuMask;
}